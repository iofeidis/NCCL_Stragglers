#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel to execute the loop
__global__ void loopKernel(int iterations) {
    for (int i = 0; i < iterations; i++) {
        printf("");
    }
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        printf("Usage: %s <number_of_iterations> <number_of_runs>\n", argv[0]);
        return 1;
    }

    int iterations = atoi(argv[1]);
    int runs = atoi(argv[2]);

    // Initialize CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Warm-up run
    loopKernel<<<1, 1>>>(iterations);
    hipDeviceSynchronize();

    float totalMilliseconds = 0;

    for (int i = 0; i < runs; i++) {
        // Record the start event
        hipEventRecord(start);

        // Launch the kernel
        loopKernel<<<1, 1>>>(iterations);

        // Record the stop event
        hipEventRecord(stop);

        // Synchronize and calculate the elapsed time
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        totalMilliseconds += milliseconds;
    }

    // Calculate the average time
    float averageMilliseconds = totalMilliseconds / runs;

    // Print the average elapsed time
    printf("Average time for loop with %d iterations over %d runs: %f milliseconds\n", iterations, runs, averageMilliseconds);

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}